#include "hip/hip_runtime.h"
#include "Scalers.cuh"
#include "Stopwatch.cuh"

#include <cmath>
#include "hip/hip_runtime.h"
#include ""

#include <utility>
#include <iostream>

#define HANDLE_ERROR( err ) ( HandleError( err, __FILE__, __LINE__ ) )

static void HandleError(hipError_t err, const char *file, int line)
{
    if (err != hipSuccess)
    {
        printf("%s in %s at line %d\n", hipGetErrorString(err),file, line);
        exit(EXIT_FAILURE);
    }
}

namespace
{
    using namespace std;

    const uint32_t BLOCK_DIM = 200;

	const uint32_t ROWS_AMOUNT = 20000;
	const uint32_t ATTRIBUTES_AMOUNT = 16;
    const uint32_t MATRIX_SIZE = ATTRIBUTES_AMOUNT + 1; // attributes + its class
    
    __device__ void findLocalMinMax(double* devAttributes, double* mins, double* maxes)
    {
        int thisThreadStart = threadIdx.x * ROWS_AMOUNT / blockDim.x + blockIdx.x * ROWS_AMOUNT;
        const int nextThreadStart = (threadIdx.x + 1) * ROWS_AMOUNT / blockDim.x + blockIdx.x * ROWS_AMOUNT;
        double localMin = devAttributes[thisThreadStart];
        double localMax = localMin;
        for (int row = thisThreadStart; row < nextThreadStart; ++row)
        {
            auto value = devAttributes[row];
            if (value < localMin)
            {
                localMin = value;
            }
            
            if (value > localMax)
            {
                localMax = value;
            }
        }

        mins[blockIdx.x * blockDim.x + threadIdx.x] = localMin;
        maxes[blockIdx.x * blockDim.x + threadIdx.x] = localMax;
    }

    __device__ void findMinMax(volatile double* min, volatile double* max, double* localMin, double* localMax)
    {
        if (threadIdx.x == 0)
        {
            min[blockIdx.x] = localMin[blockIdx.x * blockDim.x];
            max[blockIdx.x] = localMax[blockIdx.x * blockDim.x];
        }
        __syncthreads();

        for (int i = threadIdx.x; i < blockDim.x; ++i)
        {
        auto localMinValue = localMin[blockIdx.x * blockDim.x + i];
        // auto localMinValue = localMin[blockIdx.x * blockDim.x + threadIdx.x];
        if (min[blockIdx.x] > localMinValue)
        {
            min[blockIdx.x] = localMinValue;
        }
        auto localMaxValue = localMax[blockIdx.x * blockDim.x + i];
        // auto localMaxValue = localMax[blockIdx.x * blockDim.x + threadIdx.x];
        if (max[blockIdx.x] < localMaxValue)
        {
            max[blockIdx.x] = localMaxValue;
        }
        }
    }
    
    __global__ void normalize(double* devAttributes, double* localMin, double* localMax)
    {
        // double localMin[ATTRIBUTES_AMOUNT * BLOCK_DIM];
        // double localMax[ATTRIBUTES_AMOUNT * BLOCK_DIM];

        findLocalMinMax(devAttributes, localMin, localMax);
        __syncthreads();

        __shared__ double max[ATTRIBUTES_AMOUNT];
        __shared__ double min[ATTRIBUTES_AMOUNT];

        // if (threadIdx.x == 0)
        // {
        findMinMax(min, max, localMin, localMax);
        // }
        __syncthreads();

        // printf ("BEFORE SAVE blockIdx.x=%d threadIdx.x=%d [%lf : %lf]\n", blockIdx.x, threadIdx.x, min[blockIdx.x], max[blockIdx.x]);
        if (threadIdx.x == 0)
        {
            devAttributes[blockIdx.x] = min[blockIdx.x];
            devAttributes[20000 + blockIdx.x] = max[blockIdx.x];
        }








        // if (threadIdx.x == 0)
        // {
        //     for (int i = blockIdx.x * blockDim.x; i < (blockIdx.x + 1) * blockDim.x; ++i)
        //     {
        //         if (localMin[i] < min[blockIdx.x])
        //         {
        //             min[blockIdx.x] = localMin[i];
        //         }
        //         if (localMax[i] > max[blockIdx.x])
        //         {
        //             max[blockIdx.x] = localMax[i];
        //         }
        //     }
        //     // gridGroup.sync();

        //     if (threadIdx.x == 0)
        //     {
        //         for (int i = 0; i < 3; ++i)
        //         {
        //             // printf("%d row: min=%d; max=%d\n", i, min[i], max[i]);
        //             devAttributes[i] = min[i];
        //             devAttributes[20000 + i] = max[i];
        //         }
        //     }
        // }
    

        //double diff = minMax.first - minMax.second;
    
        //for (auto& value : attributeSet)
        //{
            //value = (value - minMax.first) / diff;
        //}
    }
}

void Scalers::normalize(vector<double>& attributesValues)
{
    double* devLocalMin;
    HANDLE_ERROR(hipMalloc(&devLocalMin, ATTRIBUTES_AMOUNT * BLOCK_DIM * sizeof(double)));
    double* devLocalMax;
    HANDLE_ERROR(hipMalloc(&devLocalMax, ATTRIBUTES_AMOUNT * BLOCK_DIM * sizeof(double)));
    

    double* attributes = attributesValues.data();
	double* devAttributes = nullptr;
	HANDLE_ERROR(hipMalloc(&devAttributes, attributesValues.size() * sizeof(double)));
    HANDLE_ERROR(hipMemcpy(devAttributes, attributes, attributesValues.size() * sizeof(double), hipMemcpyHostToDevice));
    // Stopwatch watch;
    // watch.start();
	::normalize<<<ATTRIBUTES_AMOUNT, BLOCK_DIM>>>(devAttributes, devLocalMin, devLocalMax);
    // watch.stop();
    // watch.displayTime();
	HANDLE_ERROR(hipMemcpy(attributes, devAttributes, attributesValues.size() * sizeof(double), hipMemcpyDeviceToHost));
    // for (int i = 0; i < ATTRIBUTES_AMOUNT; ++i)
    // {
    //     cout << "j = " << i << endl;
    //     HANDLE_ERROR(hipMemcpy(&attributes[i], &devAttributes[i], ROWS_AMOUNT * sizeof(double), hipMemcpyDeviceToHost));
    //     HANDLE_ERROR(hipFree(&devAttributes[i]));
    // }
    hipFree(devAttributes);
    

    
    double localMax[ATTRIBUTES_AMOUNT * BLOCK_DIM];
	HANDLE_ERROR(hipMemcpy(localMax, devLocalMax, ATTRIBUTES_AMOUNT * BLOCK_DIM * sizeof(double), hipMemcpyDeviceToHost));
    hipFree(devLocalMax);
    double localMin[ATTRIBUTES_AMOUNT * BLOCK_DIM];
	HANDLE_ERROR(hipMemcpy(localMin, devLocalMin, ATTRIBUTES_AMOUNT * BLOCK_DIM * sizeof(double), hipMemcpyDeviceToHost));
    hipFree(devLocalMin);

    // for (int i = 0; i < ATTRIBUTES_AMOUNT; ++i)
    // {
    //     cout << " localMin[" << i <<"] ";
    //     for (int j = 0; j < BLOCK_DIM; ++j)
    //     {
    //         cout << localMin[i*BLOCK_DIM + j] << " ";
    //     }
    //     cout << endl;
    //     cout << " localMax[" << i <<"] ";
    //     for (int j = 0; j < BLOCK_DIM; ++j)
    //     {
    //         cout << localMax[i*BLOCK_DIM + j] << " ";
    //     }
    //     cout << endl;
    // }
}

// vector<double*> Scalers::transformToRawPointer(vector<vector<double>>& matrix)
// {
//     // vector<double*> rawMatrix;
//     // int i = 0;
// 	// for (auto& column : matrix)
// 	// {
//     //     double* a = new double[ROWS_AMOUNT];
//     //     for (int j = 0; j < ROWS_AMOUNT; ++j)
//     //     {
//     //         a[j] = matrix[i][j];
//     //     }
//     //     rawMatrix.push_back(a);
//     //     ++i;
// 	// }
// 	// return rawMatrix.data();

//     // double** a = new double*[MATRIX_SIZE];
//     // for (int i = 0; i < MATRIX_SIZE; ++i)
//     // {
//     //     a[i] = new double[ROWS_AMOUNT];
//     //     for (int j = 0; j < ROWS_AMOUNT; ++j)
//     //     {
//     //         a[i][j] = matrix[i][j];
//     //     }
//     // }
//     // return a;

// 	vector<double*> rawMatrix;
// 	for (auto& column : matrix)
// 	{
// 		rawMatrix.push_back(column.data());
// 	}
// 	return rawMatrix;
// }

//pair<double, double> Scalers::findMinMax(vector<double> &attributeSet)
//{
//    double min = attributeSet.at(0);
//    double max = min;
//
//    for (const auto& value : attributeSet)
//    {
//        if (value < min)
//        {
//            min = value;
//        }
//        
//        if (value > max)
//        {
//            max = value;
//        }
//    }
//
//    return std::make_pair(min, max);
//}

void Scalers::standarize(vector<double> &attributeSet)
{
    const auto averageVariation = findAverageAndVariation(attributeSet);

    for (auto& value : attributeSet)
    {
        value = (value - averageVariation.first) / averageVariation.second;
    }
}

pair<double, double> Scalers::findAverageAndVariation(vector<double> &attributeSet)
{
    double average{};
    
    for (const auto& value : attributeSet)
    {
        average += value;
    }
    average /= attributeSet.size();

    double variation{};
    for (const auto& value : attributeSet)
    {
        auto tmp = value - average;
        variation += tmp * tmp;
    }
    variation /= attributeSet.size(); // variance
    variation = sqrt(variation);

    return std::make_pair(average, variation);
}
